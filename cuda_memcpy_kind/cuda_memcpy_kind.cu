#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void printData(int* _dDataPtr)
{
    printf("%d",_dDataPtr[threadIdx.x]);
}

__global__ void setData(int* _dDataPtr)
{
    _dDataPtr[threadIdx.x] = 2;
}


int main()
{
    int data[10] = {0,};
    for(int i = 0 ; i < 10 ; i++)
    {
        data[i] = 1;
    }
    int* dDataPtr;
    hipMalloc(&dDataPtr, sizeof(int) * 10);
    hipMemset(dDataPtr, 0x00, sizeof(int) * 10);
    
    printf("Data in device: ");
    printData<<<1, 10>>>(dDataPtr);

    hipMemcpy( dDataPtr, data, sizeof(int) * 10, hipMemcpyHostToDevice);
    printf("\nHost -> Devices :");
    printData<<<1, 10>>>(dDataPtr);

    setData<<<1, 10>>>(dDataPtr);

    hipMemcpy( data, dDataPtr, sizeof(int) * 10, hipMemcpyDeviceToHost);
    printf("\nDevices -> Host :");
    for(int i = 0 ; i < 10 ; i++)
    {
        printf("%d",data[i]);
    }



    hipFree( dDataPtr);


}    